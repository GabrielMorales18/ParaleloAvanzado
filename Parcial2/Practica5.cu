#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <random>
#include <stdio.h>
#include <iostream>
#include <time.h>


__global__ void transposeUnroll(int* a, int* b, int n) {
    int gid = (threadIdx.x + threadIdx.y * blockDim.x) + (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y);
    int offset = blockDim.x / 2;

    for (int i = 0; i < (n * n + blockDim.x * blockDim.y - 1) / (blockDim.x * blockDim.y); i += 2)
    {
        if (gid + blockDim.x * blockDim.y * i < n * n) {
            b[(gid % n * n + gid / n) + offset * i] = a[gid + blockDim.x * blockDim.y * i];
        }
        if (gid + blockDim.x * blockDim.y * i + blockDim.x * blockDim.y < n * n) {
            b[(gid % n * n + gid / n) + offset * i + offset] = a[gid + blockDim.x * blockDim.y * i + blockDim.x * blockDim.y];
        }
    }

}

int main() {
    const int n = 8;
    int* host_a, * host_b;
    int* dev_a, * dev_b;

    host_a = (int*)malloc(n * n * sizeof(int));
    host_b = (int*)malloc(n * n * sizeof(int));

    hipMalloc(&dev_a, n * n * sizeof(int));
    hipMalloc(&dev_b, n * n * sizeof(int));

    srand(time(NULL));

    for (int i = 0; i < n * n; i++) {
        int r1 = (rand() % (10));
        host_a[i] = r1;
        host_b[i] = 0;
    }

    printf("Original: \n");
   
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", host_a[i * n + j]);
        }
        printf("\n");
    }

    hipMemcpy(dev_a, host_a, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, n * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(4, 4);

    transposeUnroll << <1, block >> > (dev_a, dev_b, n);
    hipMemcpy(host_b, dev_b, n * n * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipDeviceReset();

    printf("Transpose: \n");

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", host_b[i * n + j]);
        }
        printf("\n");
    }

    free(host_a);
    free(host_b);
    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}
